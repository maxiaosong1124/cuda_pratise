#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define CUDA_CHECK(call)                                                     \
    do {                                                                     \
        hipError_t err__ = (call);                                          \
        if (err__ != hipSuccess) {                                          \
            fprintf(stderr, "CUDA Error %s:%d: %s\n", __FILE__, __LINE__,    \
                    hipGetErrorString(err__));                              \
            exit(EXIT_FAILURE);                                              \
        }                                                                    \
    } while (0)
template<const uint BLOCKSIZE>
__global__ void sgemm_coaliescing(const float* A, const float* B, float* C,
                            int M, int N, int K, float alpha, float beta)
{
    // the output block that we want to compute in this threadblock
    const uint cRow = blockIdx.y;  // 修正：blockIdx.y 对应行
    const uint cCol = blockIdx.x;  // 修正：blockIdx.x 对应列

    // allocate buffer for current block in fast shared mem
    // shared mem is shared between all threads in a block
    __shared__ float As[BLOCKSIZE * BLOCKSIZE];
    __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];

    // the inner row & col that we're accessing in this thread
    const uint threadCol = threadIdx.x;  // 修正：直接使用 threadIdx.x
    const uint threadRow = threadIdx.y;  // 修正：直接使用 threadIdx.y

    // advance pointers to the starting positions
    A += cRow * BLOCKSIZE * K;                    // row=cRow, col=0
    B += cCol * BLOCKSIZE;                        // row=0, col=cCol
    C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // row=cRow, col=cCol

    float tmp = 0.0;
    for (int bkIdx = 0; bkIdx < K; bkIdx += BLOCKSIZE) {
        // 边界检查并加载 A 和 B
        if (cRow * BLOCKSIZE + threadRow < M && bkIdx + threadCol < K) {
            As[threadRow * BLOCKSIZE + threadCol] = A[threadRow * K + threadCol];
        } else {
            As[threadRow * BLOCKSIZE + threadCol] = 0.0f;
        }
        
        if (bkIdx + threadRow < K && cCol * BLOCKSIZE + threadCol < N) {
            Bs[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];
        } else {
            Bs[threadRow * BLOCKSIZE + threadCol] = 0.0f;
        }

        // block threads in this block until cache is fully populated
        __syncthreads();
        A += BLOCKSIZE;
        B += BLOCKSIZE * N;

        // execute the dotproduct on the currently cached block
        for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
            tmp += As[threadRow * BLOCKSIZE + dotIdx] *
                   Bs[dotIdx * BLOCKSIZE + threadCol];
        }
        // need to sync again at the end, to avoid faster threads
        // fetching the next block into the cache before slower threads are done
        __syncthreads();
    }
    
    // 边界检查并写回结果
    if (cRow * BLOCKSIZE + threadRow < M && cCol * BLOCKSIZE + threadCol < N) {
        C[threadRow * N + threadCol] =
            alpha * tmp + beta * C[threadRow * N + threadCol];
    }
}

void sgemm_cpu(const float* A, const float* B, float* C, int M, int N, int K, float alpha, float beta)
{
    for(int m = 0; m < M; ++m)
    {
        for(int n = 0; n < N; ++n)
        {
            float tmp = 0;
            for(int k = 0; k < K; ++k)
            {
                tmp += A[m * K + k] * B[k * N + n];
            }
            C[m * N + n] = alpha * tmp + beta * C[m * N + n];
        }
    }
}

static void init_matrix(float* ptr, size_t size, unsigned seed, float scale)
{
    srand(seed);
    for(size_t i = 0; i < size; ++i)
    {
        // Simple pseudo-random values in [0, scale)
        ptr[i] = scale * (float)rand() / (float)RAND_MAX;
    }
}

static size_t compare_results(const float* ref, const float* got, size_t size,
                              float tol, float* out_max_abs_err)
{
    double max_abs = 0.0;
    size_t mismatches = 0;
    for(size_t i = 0; i < size; ++i)
    {
        double diff = (double)got[i] - (double)ref[i];
        double abd = fabs(diff);
        if (abd > max_abs) max_abs = abd;
        if (abd > (double)tol) ++mismatches;
    }
    *out_max_abs_err = (float)max_abs;
    return mismatches;
}

int main(int argc, char** argv)
{
    // Problem size (M x K) * (K x N) = (M x N)
    constexpr uint BLOCKSIZE = 32;
    int M = 256;
    int N = 256;
    int K = 256;
    if (argc == 4) {
        M = atoi(argv[1]);
        N = atoi(argv[2]);
        K = atoi(argv[3]);
    }
    const float alpha = 1.0f;
    const float beta  = 0.0f; // set beta=0 for a simpler check

    size_t sizeA = (size_t)M * (size_t)K;
    size_t sizeB = (size_t)K * (size_t)N;
    size_t sizeC = (size_t)M * (size_t)N;

    float *hA = (float*)malloc(sizeA * sizeof(float));
    float *hB = (float*)malloc(sizeB * sizeof(float));
    float *hC = (float*)malloc(sizeC * sizeof(float));
    float *hC_ref = (float*)malloc(sizeC * sizeof(float));
    if (!hA || !hB || !hC || !hC_ref) {
        fprintf(stderr, "Host allocation failed\n");
        return EXIT_FAILURE;
    }

    init_matrix(hA, sizeA, 123u, 1.0f);
    init_matrix(hB, sizeB, 456u, 1.0f);
    init_matrix(hC, sizeC, 789u, 1.0f); // initial C for beta term
    // Copy initial C to reference buffer
    for (size_t i = 0; i < sizeC; ++i) hC_ref[i] = hC[i];

    float *dA = nullptr, *dB = nullptr, *dC = nullptr;
    CUDA_CHECK(hipMalloc((void**)&dA, sizeA * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dB, sizeB * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&dC, sizeC * sizeof(float)));

    CUDA_CHECK(hipMemcpy(dA, hA, sizeA * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, hB, sizeB * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dC, hC, sizeC * sizeof(float), hipMemcpyHostToDevice));

    // Use 2D block layout for shared memory version
    dim3 block(BLOCKSIZE, BLOCKSIZE);
    dim3 grid((N + BLOCKSIZE - 1) / BLOCKSIZE, (M + BLOCKSIZE - 1) / BLOCKSIZE);

    // Optional timing
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    sgemm_coaliescing<32><<<grid, block>>>(dA, dB, dC, M, N, K, alpha, beta);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float ms = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    CUDA_CHECK(hipMemcpy(hC, dC, sizeC * sizeof(float), hipMemcpyDeviceToHost));

    // CPU reference
    sgemm_cpu(hA, hB, hC_ref, M, N, K, alpha, beta);

    const float tol = 5e-3f; // 0.005
    float max_abs_err = 0.0f;
    size_t mismatches = compare_results(hC_ref, hC, sizeC, tol, &max_abs_err);

    printf("SGEMM shared memory GPU vs CPU\n");
    printf("Dims: M=%d N=%d K=%d, block=(%d,%d) grid=(%d,%d)\n",
           M, N, K, block.x, block.y, grid.x, grid.y);
    printf("Kernel time: %.3f ms\n", ms);
    printf("Abs tolerance: %.6f\n", tol);
    printf("Max abs error: %.6e\n", max_abs_err);
    printf("Mismatches: %zu / %zu\n", mismatches, sizeC);

    bool ok = (mismatches == 0);
    printf("Result: %s\n", ok ? "PASS" : "FAIL");

    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dC));
    free(hA); free(hB); free(hC); free(hC_ref);

    return ok ? EXIT_SUCCESS : EXIT_FAILURE;
}