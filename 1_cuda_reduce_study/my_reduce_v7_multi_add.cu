#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256

__device__ void warp_reduce(volatile float* cache, unsigned int tid)
{
        cache[tid] += cache[tid + 32];
        cache[tid] += cache[tid + 16];
        cache[tid] += cache[tid + 8];
        cache[tid] += cache[tid + 4];
        cache[tid] += cache[tid + 2];
        cache[tid] += cache[tid + 1];
}
template<unsigned int NUM_PER_BLOCK>
__global__ void reduce(float* d_input, float* d_output)
{
    __shared__ float shared[THREAD_PER_BLOCK];
    float *input_begin = d_input + blockIdx.x * NUM_PER_BLOCK; 
    shared[threadIdx.x] = 0.0f;

    for(int i = 0; i < NUM_PER_BLOCK / THREAD_PER_BLOCK; ++i)
    {
        int idx = threadIdx.x + i * THREAD_PER_BLOCK;
        if (idx < NUM_PER_BLOCK) 
        {
            shared[threadIdx.x] += input_begin[idx];
        }
    }
    __syncthreads();//这里的syncthreads在循环之外是因为所有的线程都是独立操作共享内存的地址的，没有线程之间的数据依赖

    //1.使用宏对循环进行完全展开
    #pragma unroll
    for(int i = blockDim.x / 2; i > 32; i /= 2)
    {
        if(threadIdx.x < i)
        {
            shared[threadIdx.x] += shared[threadIdx.x + i];
        }
      __syncthreads();//这里在循环内部，是一位在线程操作的时候要依赖于不同的线程操作不同的共享内存地址，需要保证它们两个操作完毕才能进行下一步，否则会导致数据出现错误
    }
      
    //2.手动展开循环
    // if(THREAD_PER_BLOCK >= 512) //为了支持更大的线程块
    // {
    //     if(threadIdx.x < 256)
    //     {
    //         shared[threadIdx.x] += shared[threadIdx.x + 256];
    //     }
    //     __syncthreads();
    // }

    // if(THREAD_PER_BLOCK >= 256)
    // {
    //     if(threadIdx.x < 128)
    //     {
    //         shared[threadIdx.x] += shared[threadIdx.x + 128];
    //     }
    //     __syncthreads();
    // }

    // if(THREAD_PER_BLOCK >= 128)
    // {
    //     if(threadIdx.x < 64)
    //     {
    //         shared[threadIdx.x] += shared[threadIdx.x + 64];
    //         __syncthreads();
    //     }
    // }

    if(threadIdx.x < 32)
    {
        warp_reduce(shared, threadIdx.x);
    }

    if(threadIdx.x == 0)
    {
        d_output[blockIdx.x] = shared[0];
    }
}

bool check_result(float* result, float* output, int n)
{
    for(int i = 0; i < n; ++i)
    {
        if(abs(result[i] - output[i]) > 0.005)
        {
            printf("The ans is wrong!\n");
            printf("The result is at index %d: %f, but the output is %f\n", i, result[i], output[i]);
            return false;
        }
    }
    printf("The ans is right!\n");
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;

    float* input = (float*) malloc (N * sizeof(float));
    float* d_input = nullptr;
    hipMalloc((void**)&d_input, N * sizeof(float));

    const int block_num = 1024;
    const int num_per_block = N / block_num;

    float* output = (float*)malloc(block_num * sizeof(float));
    float* d_output = nullptr;
    hipMalloc((void**)&d_output, block_num * sizeof(float));
    float* result = (float*) malloc(block_num * sizeof(float));
    
    for(int i = 0; i < N; ++i)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    //cpu_calc
    for(int i = 0; i < block_num; ++i)
    {
        float cur = 0;
        for(int j = 0; j < num_per_block; ++j)
        {
            cur += input[i * num_per_block + j];
        }
        result[i] = cur;
    }

    //copy input data to device
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(block_num);
    dim3 block(THREAD_PER_BLOCK);

    //launch kernel
    printf("kernel start\n");
    reduce<num_per_block><<<grid, block>>>(d_input, d_output);
    printf("kernel end\n");

    //copy output data to host
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    //check result
    bool is_correct = check_result(result, output, block_num);
    if (is_correct) {
        printf("All results are correct!\n");
    } else {
        printf("There are errors in the results.\n");
    }


    //free memory
    free(input);
    free(output);
    free(result);

    hipFree(d_input);
    hipFree(d_output);


    return 0;
}
