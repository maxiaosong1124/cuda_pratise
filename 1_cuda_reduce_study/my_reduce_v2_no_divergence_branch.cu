#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstdio>


#define THREAD_PER_BLOCK 256

__global__ void reduce(float* d_input, float* d_output)
{   
    
    //2.按照GPU的线程索引来进行实现
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ float shared[THREAD_PER_BLOCK];
    shared[threadIdx.x] = d_input[tid];
    __syncthreads();
    for(int i = 1; i < blockDim.x; i *= 2)
    {
        if(threadIdx.x < blockDim.x /(i * 2))
        {
            int index = threadIdx.x * 2 * i;
            shared[index] += shared[index + i];
        }
        __syncthreads();
    }

    if(threadIdx.x == 0)
    {
        d_output[blockIdx.x] = shared[0];
    }


    //1.按照block内的线程偏移地址来进行实现，类似于CPU的计算方式,这里是使用了连续的线程来进行计算，与之前的v1中的在进入后续迭代之后会有闲置线程不同，也可以减少分支分歧
    // __shared__ float shared[THREAD_PER_BLOCK];

    // float* input_begin = d_input + blockDim.x * blockIdx.x;
    // shared[threadIdx.x] = input_begin[threadIdx.x];
    // __syncthreads();
    // for(int i = 1; i < blockDim.x; i *= 2)
    // {
    //     if(threadIdx.x < blockDim.x / (i * 2))
    //     {
    //         //input_begin[threadIdx.x] += input_begin[threadIdx.x + i];
    //         int index = threadIdx.x * 2 * i;
    //         shared[index] += shared[index + i];
    //     }
    //     __syncthreads();
    // }

    // if(threadIdx.x == 0)
    // {
    //     d_output[blockIdx.x] = shared[0];
    // }
}

bool check(float* output, float* result, int n)
{
    for(int i = 0; i < n; ++i)
    {
        if(abs(output[i] - result[i]) > 0.005)
        {
            printf("Error at index %d: output = %f, result = %f\n", i, output[i], result[i]);
            return false;
        }
    }
    return true;
}

int main()
{
    constexpr int N = 32 * 1024 * 1024;
    float* input = (float*)malloc(N * sizeof(float));
    float* d_input;
    hipMalloc((void**)&d_input, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK;
    float* output = (float*)malloc(N / THREAD_PER_BLOCK * sizeof(float));
    float* d_output;
    hipMalloc((void**)&d_output, N / THREAD_PER_BLOCK * sizeof(float));
    float* result = (float*)malloc(N / THREAD_PER_BLOCK * sizeof(float));

    //初始化
    for(int i = 0; i < N; ++i)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    //cpu计算结果
    for(int i = 0; i < block_num; ++i)
    {
        float cur = 0;
        for(int j = 0; j < THREAD_PER_BLOCK; ++j)
        {
            cur += input[i * THREAD_PER_BLOCK + j];
        }
        result[i] = cur;
    }

    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);
    dim3 grid(N / THREAD_PER_BLOCK);
    dim3 block(THREAD_PER_BLOCK);

    reduce<<<grid, block>>>(d_input, d_output);

    hipMemcpy(output, d_output, N / THREAD_PER_BLOCK * sizeof(float), hipMemcpyDeviceToHost);
    if(check(output, result, block_num))
    {
        printf("The result is right\n");
    }
    else
    {
        printf("The result is wrong\n");
    }

    free(input);
    free(output);

    free(result);
    hipFree(d_input);
    hipFree(d_output);
    hipDeviceReset();

    return 0;
}