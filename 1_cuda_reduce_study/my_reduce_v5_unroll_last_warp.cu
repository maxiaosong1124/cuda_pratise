#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256

__device__ void warp_reduce(volatile float* cache, unsigned int tid)
{
    cache[tid] += cache[tid + 32];
    cache[tid] += cache[tid + 16];
    cache[tid] += cache[tid + 8];
    cache[tid] += cache[tid + 4];
    cache[tid] += cache[tid + 2];
    cache[tid] += cache[tid + 1];
}

__global__ void reduce(float* d_input, float* d_output)
{
    // __shared__ float shared[THREAD_PER_BLOCK];

    // float* input_begin = d_input + blockDim.x * blockIdx.x * 2;
    // shared[threadIdx.x]  = input_begin[threadIdx.x] + input_begin[threadIdx.x + blockDim.x];
    // __syncthreads();

    // for(int i = blockDim.x / 2; i > 32; i /= 2)
    // {
    //     if(threadIdx.x < i)
    //     {
    //         shared[threadIdx.x] += shared[threadIdx.x + i];
    //     }
    //     __syncthreads();
    // }

    // if(threadIdx.x < 32)
    // {
    //     warp_reduce(shared, threadIdx.x);
    // }

    // if(threadIdx.x == 0)
    // {
    //     d_output[blockIdx.x] = shared[0];
    // }
    //2.使用全局索引tid进行计算
    __shared__ float shared[THREAD_PER_BLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x * 2;
    shared[threadIdx.x] = d_input[tid] + d_input[tid + blockDim.x];
    __syncthreads();
    for(int i = blockDim.x / 2; i > 32; i /= 2)
    {
        if(threadIdx.x < i)
        {
            shared[threadIdx.x] += shared[threadIdx.x + i];
        }
        __syncthreads();
    }

    if(threadIdx.x < 32)
    {
        warp_reduce(shared, threadIdx.x);
    }

    if(threadIdx.x == 0)
    {
        d_output[blockIdx.x] = shared[0];
    }

}

bool check_result(float* result, float* output, int n)
{
    for(int i = 0; i < n; ++i)
    {
        if(abs(result[i] - output[i]) > 0.005)
        {
            printf("The ans is wrong!\n");
            printf("The result is at index %d: %f, but the output is %f\n", i, result[i], output[i]);
            return false;
        }
    }
    printf("The ans is right!\n");
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;

    float* input = (float*) malloc (N * sizeof(float));
    float* d_input = nullptr;
    hipMalloc((void**)&d_input, N * sizeof(float));

    int block_num = N / THREAD_PER_BLOCK / 2;
    float* output = (float*)malloc(block_num * sizeof(float));
    float* d_output = nullptr;
    hipMalloc((void**)&d_output, block_num * sizeof(float));
    float* result = (float*) malloc(block_num * sizeof(float));
    
    for(int i = 0; i < N; ++i)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    //cpu_calc
    for(int i = 0; i < block_num; ++i)
    {
        float cur = 0;
        for(int j = 0; j < 2 * THREAD_PER_BLOCK; ++j)
        {
            cur += input[i * 2 * THREAD_PER_BLOCK + j];
        }
        result[i] = cur;
    }

    //copy input data to device
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(block_num);
    dim3 block(THREAD_PER_BLOCK);

    //launch kernel
    reduce<<<grid, block>>>(d_input, d_output);

    //copy output data to host
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    //check result
    bool is_correct = check_result(result, output, block_num);
    if (is_correct) {
        printf("All results are correct!\n");
    } else {
        printf("There are errors in the results.\n");
    }


    //free memory
    free(input);
    free(output);
    free(result);

    hipFree(d_input);
    hipFree(d_output);


    return 0;
}
