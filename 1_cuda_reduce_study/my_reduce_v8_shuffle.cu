#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define THREAD_PER_BLOCK 256

__device__ float warp_reduce_shuffle(float val)
{
    // 使用 warp shuffle 指令进行 warp 内的规约
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}
template<unsigned int NUM_PER_BLOCK>
__global__ void reduce(float* d_input, float* d_output)
{
    __shared__ float shared[THREAD_PER_BLOCK];
    float *input_begin = d_input + blockIdx.x * NUM_PER_BLOCK; 
    shared[threadIdx.x] = 0.0f;

    for(int i = 0; i < NUM_PER_BLOCK / THREAD_PER_BLOCK; ++i)
    {
        int idx = threadIdx.x + i * THREAD_PER_BLOCK;
        if (idx < NUM_PER_BLOCK) {
            shared[threadIdx.x] += input_begin[idx];
        }
    }
    __syncthreads();

    //1.使用宏对循环进行完全展开
    #pragma unroll
    for(int i = blockDim.x / 2; i > 32; i /= 2)
    {
        if(threadIdx.x < i)
        {
            shared[threadIdx.x] += shared[threadIdx.x + i];
        }
        __syncthreads();
    }
    //2.手动展开循环
    // if(THREAD_PER_BLOCK >= 512) //为了支持更大的线程块
    // {
    //     if(threadIdx.x < 256)
    //     {
    //         shared[threadIdx.x] += shared[threadIdx.x + 256];
    //     }
    //     __syncthreads();
    // }

    // if(THREAD_PER_BLOCK >= 256)
    // {
    //     if(threadIdx.x < 128)
    //     {
    //         shared[threadIdx.x] += shared[threadIdx.x + 128];
    //     }
    //     __syncthreads();
    // }

    // if(THREAD_PER_BLOCK >= 128)
    // {
    //     if(threadIdx.x < 64)
    //     {
    //         shared[threadIdx.x] += shared[threadIdx.x + 64];
    //     }
    //     __syncthreads();
    // }

    // 使用 warp shuffle 进行 warp 内的规约
    float val = 0.0f;
    if(threadIdx.x < 32)
    {
        // 需要包含所有32个元素的和
        val = shared[threadIdx.x] + shared[threadIdx.x + 32];
        val = warp_reduce_shuffle(val);
    }

    if(threadIdx.x == 0)
    {
        d_output[blockIdx.x] = val;
    }
}

bool check_result(float* result, float* output, int n)
{
    for(int i = 0; i < n; ++i)
    {
        if(abs(result[i] - output[i]) > 0.005)
        {
            printf("The ans is wrong!\n");
            printf("The result is at index %d: %f, but the output is %f\n", i, result[i], output[i]);
            return false;
        }
    }
    printf("The ans is right!\n");
    return true;
}

int main()
{
    const int N = 32 * 1024 * 1024;

    float* input = (float*) malloc (N * sizeof(float));
    float* d_input = nullptr;
    hipMalloc((void**)&d_input, N * sizeof(float));

    const int block_num = 1024;
    const int num_per_block = N / block_num;

    float* output = (float*)malloc(block_num * sizeof(float));
    float* d_output = nullptr;
    hipMalloc((void**)&d_output, block_num * sizeof(float));
    float* result = (float*) malloc(block_num * sizeof(float));
    
    for(int i = 0; i < N; ++i)
    {
        input[i] = 2.0 * (float)drand48() - 1.0;
    }

    //cpu_calc
    for(int i = 0; i < block_num; ++i)
    {
        float cur = 0;
        for(int j = 0; j < num_per_block; ++j)
        {
            cur += input[i * num_per_block + j];
        }
        result[i] = cur;
    }

    //copy input data to device
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 grid(block_num);
    dim3 block(THREAD_PER_BLOCK);

    //launch kernel
    printf("kernel start\n");
    reduce<num_per_block><<<grid, block>>>(d_input, d_output);
    printf("kernel end\n");

    //copy output data to host
    hipMemcpy(output, d_output, block_num * sizeof(float), hipMemcpyDeviceToHost);

    //check result
    bool is_correct = check_result(result, output, block_num);
    if (is_correct) {
        printf("All results are correct!\n");
    } else {
        printf("There are errors in the results.\n");
    }


    //free memory
    free(input);
    free(output);
    free(result);

    hipFree(d_input);
    hipFree(d_output);


    return 0;
}
